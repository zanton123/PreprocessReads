

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <zlib.h>

int hNumberOfReads = 0;                 // holds number of reads that are processed
int hReadsWritten = 0;                  // number of reads written to output file
char input_filename[400]="/home/linux/cuda-workspace/PreprocessReads/test.txt"; // input file name
char output_filename[400]="/mnt/sdc/Sequences/out.txt";                         // output file name
char GZIP_PIPE[410] = "gzip -> ";       // option -gz will fallback to uncompressed if error on pipe opening
char PIGZ_PIPE[410] = "pigz -> ";       // option -pigz for multithreaded compression will fallback to gz and then to uncompressed
gzFile inputFile;
gzFile templateFile;						// holds a second pointer in input read file for templating output with read name sequence and quality
FILE *outputFile;
bool overwrite_outputfile = false;      // if set true the output file will be overwritten (cmd line option -f)
bool write_gz_output = false;           // specify .gz file compressed output (cmd line option -gz)
bool write_pigz_output = false;         // specify pigz multithreaded compression for output (-pigz option)

bool AdaptorTrimming = true;			// algorithm selectors
bool QualityTrimming = true;
bool gRNATrimming = false;
int gRNA_start;							// holds gRNA position relative to adaptor
int gRNA_length;
bool verbose = false;					// if true print details of processing to screen
bool FastQ_WARNING = false;				// set to true is a FastQ file warning has been issued

short MAX_READ_LENGTH = 203;            // maximal length of reads in input file
short MIN_OUT_READ_LEN = 26;            // minimum length of reads after trimming to write to output file

short MIN_BASE_QUAL = 20;				// parameter for minimum quality for good base
short MIN_QUAL_AFTER_BAD = 25;			// parameter for average base quality after bad base
short PHRED = 33;			        	// subtract from qualities for numerical value of solexa
bool auto_phred_detect = false;         // if true try to detect PHRED format in FastQ input file

int MAX_READS_ON_GPU = 10000000;        // maximum number of reads that fit simultaneously on GPU

#define MAX_ADAPTOR_LENGTH 100          // maximum length of adaptor sequence for trimming, can contain N for any base
#define MAX_FASTQ_BUF_LINE 500          // structure of buffer for read from FastQ file

char Adaptor[MAX_ADAPTOR_LENGTH] = "ATTAGCTAGCTAGATGATCGACTAGCT"; // adaptor on CPU side from parsing command line
short mismatches = 2;					// holds maximum number of mismatches on cpu side

int numGPUs = 0;                        // holds number of CUDA devices in the system [1 TITAN to max 2 TITAN Z]
#define MAX_GPU_NUM 4
int device[MAX_GPU_NUM];                // device numbers to use
hipStream_t upload_stream[MAX_GPU_NUM];    // cuda streams per device for overlapped asynchronous operation
hipStream_t compute_stream[MAX_GPU_NUM];
hipStream_t download_stream[MAX_GPU_NUM];

char *dSequenceArray[MAX_GPU_NUM];	    // device pointer to data array, will hold hReadSequence and hQualstr
char *dQualArray[MAX_GPU_NUM];			// device pointer to data array, will hold hReadSequence and hQualstr
short *dIntArray[MAX_GPU_NUM][2];       // two device pointers to integer array, will hold hReadLen
hipEvent_t QUAL_COPY_DONE[MAX_GPU_NUM];   // events for device host synchronization
hipEvent_t SEQ_COPY_DONE[MAX_GPU_NUM];
hipEvent_t TrimmQ_DONE[MAX_GPU_NUM];
hipEvent_t TrimmA_DONE[MAX_GPU_NUM];
hipEvent_t DTH_COPY_DONE[MAX_GPU_NUM];

short *hReadLen[MAX_GPU_NUM][2];        // two buffer to hold read length on host
char *hReadSequence[MAX_GPU_NUM][2];       // buffer holding sequences of reads on host
char *hReadQualstr[MAX_GPU_NUM];     // two buffer holding qualities of reads on host

struct read
{
    char name[MAX_FASTQ_BUF_LINE];
    char sequence[MAX_FASTQ_BUF_LINE];
    char descr[MAX_FASTQ_BUF_LINE];
    char qualstr[MAX_FASTQ_BUF_LINE];
} Read;                                 // buffer one read from FastQ file i 4 lines



// macro checks return value of the CUDA runtime call and exits the application if the call failed.

#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }


// For trimming adaptor sequences store adaptor and length as well as quality cutoffs in device constant memory

__constant__ __device__ char ADAPTOR[MAX_ADAPTOR_LENGTH];     // adaptor sequence can contain N for any match
__constant__ __device__ short ADAPTOR_LEN;                    // length of adaptor sequence in bases
__constant__ __device__ short MAX_MM=2;                       // maximum mismatches allowed not including Ns
__constant__ __device__ short QUAL_PADDING=33;				  // hold PHRED base for qualities
__constant__ __device__ short MIN_QUAL=20;					  // hold minimal quality that is considered good base
__constant__ __device__ short MIN_AVERAGE_QUAL_AFTER_BAD=25;  // minimum quality average of remaining read after bad base

void SetAdaptorSequence(const char *AdaptorSequence, const short mismatches, const short PHRED, const short MIN_BASE_QUAL, const short MIN_QUAL_AFTER_BAD)
{																// define the adaptor, mismatches and quality parameters before trimming
	short AdLen = strnlen(AdaptorSequence, MAX_ADAPTOR_LENGTH); // get length of adaptor sequence

	for(int dev=0; dev < numGPUs; dev++)
	{
		hipSetDevice(dev);
		if(verbose) printf("[%d] Setting Adaptor sequence %s, length %d on GPU \n", dev, AdaptorSequence, AdLen);
		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(ADAPTOR), AdaptorSequence, AdLen));           // copy to device
		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(ADAPTOR_LEN), &AdLen, sizeof(short)));        // constant memory
		if(verbose) printf("[%d] Setting mismatches %d, PHRED+%d on GPU \n", dev, mismatches, PHRED);
		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MAX_MM), &mismatches, sizeof(short)));
		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(QUAL_PADDING), &PHRED, sizeof(short)));
		if(verbose) printf("[%d] Setting minimum base quality %d, average read quality after bad %d on GPU \n", dev, MIN_BASE_QUAL, MIN_QUAL_AFTER_BAD);
		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MIN_QUAL), &MIN_BASE_QUAL, sizeof(short)));
		CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MIN_AVERAGE_QUAL_AFTER_BAD), &MIN_QUAL_AFTER_BAD, sizeof(short)));
	}
}


__global__ void Find_gRNA(char *dDataArray, short *read_length, int reads, short MAX_READ_LENGTH)
{
  for(long idx = blockIdx.x * blockDim.x + threadIdx.x; idx < reads; idx += blockDim.x * gridDim.x)
  {
      if (idx < reads)
      {
          char *sequence = &dDataArray[idx*MAX_READ_LENGTH];
          short rl=read_length[idx];
          short n;
          short p=0;
          short mm=0;
          read_length[idx]=0;
          while (p <= rl-ADAPTOR_LEN)
          {
              mm=0;
              for (n=0; n<ADAPTOR_LEN; n++)
              {
                  if (ADAPTOR[n]!='N' && sequence[p+n]!=ADAPTOR[n])
                  {
                     mm++;
                     if (mm > MAX_MM) break;
                  }
              }
              if (mm <= MAX_MM)
              {
                  read_length[idx]=p;
                  break;
              }
              p++;
          }
          if (mm <= MAX_MM ) continue;
          p=rl-ADAPTOR_LEN+1;
          if (rl >= ADAPTOR_LEN-1)
          {
              mm=0;
              for (n=0; n<ADAPTOR_LEN-1; n++)
              {
                  if (ADAPTOR[n]!='N' && sequence[p+n]!=ADAPTOR[n])
                  {
                     mm++;
                     if (mm >= MAX_MM) break;
                  }
              }
              if (mm < MAX_MM)
              {
                  read_length[idx]=p;
                  continue;
              }
          }
          p++;
          if (rl >= ADAPTOR_LEN-2)
          {
              mm=0;
              for (n=0; n<ADAPTOR_LEN-2; n++)
              {
                  if (ADAPTOR[n]!='N' && sequence[p+n]!=ADAPTOR[n])
                  {
                     mm++;
                     break;
                  }
              }
              if (MAX_MM > (mm+1))
              {
                  read_length[idx]=p;
              }
          }
      }
  }
}

__global__ void TrimmAdaptors(char *dDataArray, short *read_length, int reads, short MAX_READ_LENGTH)
{
  for(long idx = blockIdx.x * blockDim.x + threadIdx.x; idx < reads; idx += blockDim.x * gridDim.x)
  {
      if (idx < reads)
      {
          char *sequence = &dDataArray[idx*MAX_READ_LENGTH];
          short rl=read_length[idx];
          short n;
          short p=0;
          short mm=0;
          while (p <= rl-ADAPTOR_LEN)
          {
              mm=0;
              for (n=0; n<ADAPTOR_LEN; n++)
              {
                  if (ADAPTOR[n]!='N' && sequence[p+n]!=ADAPTOR[n])
                  {
                     mm++;
                     if (mm > MAX_MM) break;
                  }
              }
              if (mm <= MAX_MM)
              {
                  read_length[idx]=p;
                  break;
              }
              p++;
          }
          if (mm <= MAX_MM ) continue;
          p=rl-ADAPTOR_LEN+1;
          if (rl >= ADAPTOR_LEN-1)
          {
              mm=0;
              for (n=0; n<ADAPTOR_LEN-1; n++)
              {
                  if (ADAPTOR[n]!='N' && sequence[p+n]!=ADAPTOR[n])
                  {
                     mm++;
                     if (mm >= MAX_MM) break;
                  }
              }
              if (mm < MAX_MM)
              {
                  read_length[idx]=p;
                  continue;
              }
          }
          p++;
          if (rl >= ADAPTOR_LEN-2)
          {
              mm=0;
              for (n=0; n<ADAPTOR_LEN-2; n++)
              {
                  if (ADAPTOR[n]!='N' && sequence[p+n]!=ADAPTOR[n])
                  {
                     mm++;
                     break;
                  }
              }
              if (MAX_MM > (mm+1))
              {
                  read_length[idx]=p;
              }
          }
      }
  }
}

__global__ void TrimmQuals(char *quals, short *read_length, int reads, short MAX_READ_LENGTH)
{
  for(long idx = blockIdx.x * blockDim.x + threadIdx.x; idx < reads; idx += blockDim.x * gridDim.x)
  {
      if (idx < reads)
      {
          char *q = &quals[idx*MAX_READ_LENGTH];
          short rl=read_length[idx];
          short i;
          for(i=0; i<rl; i++)
          {
              if ((q[i]-QUAL_PADDING)<=MIN_QUAL)
              {
                 short s=0;
                 short r=i+1;
                 while (r<rl)
                 {
                    s=s+q[r]-QUAL_PADDING;
                    r++;
                 }
                 if(s<(MIN_AVERAGE_QUAL_AFTER_BAD*(rl-i+1)))
                 {
                    read_length[idx]=i;
                    break;
                 }
              }
          }

      }
  }
}


void InitGPUs(void)
{
	if(verbose) printf("\n");
	for (int dev=0; dev<numGPUs; dev++)                         // initialize devices and allocate cuda resources
    {
    	if(verbose) printf("[%d] Initializing GPU\n", dev);
    	CUDA_CHECK_RETURN(hipSetDevice(device[dev]));
    	CUDA_CHECK_RETURN(hipMalloc((void **)&dSequenceArray[dev], MAX_READS_ON_GPU * MAX_READ_LENGTH * sizeof(char)));    // one data array
    	CUDA_CHECK_RETURN(hipMalloc((void **)&dQualArray[dev], MAX_READS_ON_GPU * MAX_READ_LENGTH * sizeof(char)));    // one data array
    	CUDA_CHECK_RETURN(hipMalloc((void **)&dIntArray[dev][0], MAX_READS_ON_GPU * sizeof(short)));  // two dIntArrays
    	CUDA_CHECK_RETURN(hipMalloc((void **)&dIntArray[dev][1], MAX_READS_ON_GPU * sizeof(short)));

    	CUDA_CHECK_RETURN(hipStreamCreate(&upload_stream[dev]));                                   // create streams on devices
    	CUDA_CHECK_RETURN(hipStreamCreate(&compute_stream[dev]));
    	CUDA_CHECK_RETURN(hipStreamCreate(&download_stream[dev]));

    	CUDA_CHECK_RETURN(hipEventCreateWithFlags(&QUAL_COPY_DONE[dev], hipEventDisableTiming || hipEventBlockingSync));  // events for device host synchronization
    	CUDA_CHECK_RETURN(hipEventCreateWithFlags(&SEQ_COPY_DONE[dev], hipEventDisableTiming));
    	CUDA_CHECK_RETURN(hipEventCreateWithFlags(&TrimmQ_DONE[dev], hipEventDisableTiming));
    	CUDA_CHECK_RETURN(hipEventCreateWithFlags(&TrimmA_DONE[dev], hipEventDisableTiming));
    	CUDA_CHECK_RETURN(hipEventCreateWithFlags(&DTH_COPY_DONE[dev], hipEventDisableTiming || hipEventBlockingSync));

        if(verbose) printf("[%d] Allocating pinned CPU memory for reads\n", dev);
        CUDA_CHECK_RETURN(hipHostMalloc((void **) &hReadLen[dev][0], MAX_READS_ON_GPU * sizeof(short), hipHostMallocDefault));   // allocate two buffers for reads length
        CUDA_CHECK_RETURN(hipHostMalloc((void **) &hReadLen[dev][1], MAX_READS_ON_GPU * sizeof(short), hipHostMallocDefault));

        CUDA_CHECK_RETURN(hipHostMalloc((void **) &hReadSequence[dev][0], MAX_READS_ON_GPU * MAX_READ_LENGTH * sizeof(char), hipHostMallocDefault));   // two buffers for reads sequence
        CUDA_CHECK_RETURN(hipHostMalloc((void **) &hReadSequence[dev][1], MAX_READS_ON_GPU * MAX_READ_LENGTH * sizeof(char), hipHostMallocDefault));

        CUDA_CHECK_RETURN(hipHostMalloc((void **) &hReadQualstr[dev], MAX_READS_ON_GPU * MAX_READ_LENGTH * sizeof(char), hipHostMallocDefault)); // one buffer for qualities

    }
}

void FreeGPUs(void)
{
    for (int dev=0; dev<numGPUs; dev++)                         // release cuda resources from all devices
    {
    	CUDA_CHECK_RETURN(hipSetDevice(dev));

    	CUDA_CHECK_RETURN(hipEventDestroy(QUAL_COPY_DONE[dev]));
    	CUDA_CHECK_RETURN(hipEventDestroy(SEQ_COPY_DONE[dev]));
    	CUDA_CHECK_RETURN(hipEventDestroy(TrimmQ_DONE[dev]));
    	CUDA_CHECK_RETURN(hipEventDestroy(TrimmA_DONE[dev]));
    	CUDA_CHECK_RETURN(hipEventDestroy(DTH_COPY_DONE[dev]));
    	CUDA_CHECK_RETURN(hipStreamDestroy(upload_stream[dev]));
    	CUDA_CHECK_RETURN(hipStreamDestroy(compute_stream[dev]));
    	CUDA_CHECK_RETURN(hipStreamDestroy(download_stream[dev]));

    	CUDA_CHECK_RETURN(hipFree(dIntArray[dev][0]));
    	CUDA_CHECK_RETURN(hipFree(dIntArray[dev][1]));
    	CUDA_CHECK_RETURN(hipFree(dSequenceArray[dev]));
    	CUDA_CHECK_RETURN(hipFree(dQualArray[dev]));

        hipHostFree(hReadLen[dev][0]);                        // free all page locked host memory
        hipHostFree(hReadLen[dev][1]);
        hipHostFree(hReadSequence[dev][0]);
        hipHostFree(hReadSequence[dev][1]);
        hipHostFree(hReadQualstr[dev]);
    }

}

int FindGPUs(void)
{
    int deviceCount;
    numGPUs = 0;
    CUDA_CHECK_RETURN(hipGetDeviceCount(&deviceCount)); // search cuda devices and identify compute 3.5 with 6 GB
    if (deviceCount == 0) return(0);

    for (int dev=0; dev<deviceCount; dev++)
    {
    	hipSetDevice(dev);
    	hipDeviceProp_t deviceProp;
    	hipGetDeviceProperties(&deviceProp, dev);

    	int driverVersion, runtimeVersion;
    	hipDriverGetVersion(&driverVersion);
    	hipRuntimeGetVersion(&runtimeVersion);

        if(((deviceProp.major == 3 && deviceProp.minor >= 5) || (deviceProp.major > 3)) && deviceProp.totalGlobalMem > 6144000000)
        {
            if(verbose)
            {
            	printf("\nDevice %d: %s\n", dev, deviceProp.name);
        	    printf("  CUDA Driver / Runtime Version:     %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
        	    printf("  CUDA Compute Capability:           %d.%d\n", deviceProp.major, deviceProp.minor);
                    printf("  PCI bus ID:                        %d:%d\n", deviceProp.pciBusID, deviceProp.pciDeviceID);
        	    printf("  Total amount of global memory:     %.0f MBytes (%llu bytes)\n", (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
        	    printf("  Number of Multiprocessors:         %d\n", deviceProp.multiProcessorCount);
            }
        	device[numGPUs] = dev;
        	numGPUs++;
        	if (numGPUs > MAX_GPU_NUM) break;
        }
    }
    return(numGPUs);
}

void OpenFiles()
{
    // handle compression using zlib gzFile functions for input and pipe gzip for output

    if(!overwrite_outputfile)       // test if output file exists if option -f is not specified
    {
        outputFile = fopen(output_filename, "r");
    	if(outputFile!=NULL)
    	{
    		printf("ERROR output file exists already. To overwrite run with option -f\n");
    		exit(255);
    	}  // do not attempt close file as we would have already exited if opening was successful !
    }
	inputFile = gzopen(input_filename, "r");
    if(inputFile == NULL)
    {
    	printf("ERROR opening input file: %s\n", input_filename);
    	exit(255);
    }
	templateFile = gzopen(input_filename, "r");  // holds a second access to input read file for rereading positions for output
    if(templateFile == NULL)
    {
    	printf("ERROR opening second handle to input file.\n");
    	exit(255);
    }
    if(write_pigz_output)
    {
    	outputFile = popen(strcat(PIGZ_PIPE, output_filename), "w");
        if(outputFile == NULL)
        {
        	printf("Option -pigz: could not open pipe ... falling back on libz.\n\tMake sure pigz is installed with: sudo apt-get install pigz\n");
        	write_gz_output = true;
        	write_pigz_output = false;
        }
    }
	if(write_gz_output && !write_pigz_output)
	{
		outputFile = popen(strcat(GZIP_PIPE, output_filename), "w");
        if(outputFile == NULL)
        {
        	printf("Option -gz: could not open pipe ... falling back to uncompressed output.\n");
        	write_gz_output = false;
        }
	}
	if(!write_gz_output && !write_pigz_output) outputFile = fopen(output_filename, "w");
    if(outputFile == NULL)
    {
    	printf("ERROR opening output file: %s\n", output_filename);
    	exit(255);
    }
}

int ReadBlockOfReads(int max_num_to_read, short *hReadLen, char* hReadSequence, char *hReadQualstr)
{
	int n = 0;                                                                    // returns the number of reads read
	short length;

    while(gzgets(inputFile, Read.name, MAX_FASTQ_BUF_LINE) != NULL)
    {
    	if(Read.name[0] == 13) break;				// end if CR <13> or LF <10> are encountered
    	if(Read.name[0] == 10) break;
    	if(gzgets(inputFile, Read.sequence, MAX_FASTQ_BUF_LINE)==NULL)
        {
            printf("ERROR parsing input!\n");
            exit(255);
        }
        if(gzgets(inputFile, Read.descr, MAX_FASTQ_BUF_LINE)==NULL)
        {
            printf("ERROR parsing input!\n");
            exit(255);
        }
        if(gzgets(inputFile, Read.qualstr, MAX_FASTQ_BUF_LINE)==NULL)
        {
            printf("ERROR parsing input!\n");
            exit(255);
        }

        if(!FastQ_WARNING)                         // write warning if FastQ file contains non-standard lines
        {
            if(Read.name[0]!='@' || Read.descr[0]!='+')
            {
            	printf("Warning: Encountered unusual line identifiers in FastQ file format.\n");
            	FastQ_WARNING = true;
            }
        }
        length = strlen(Read.sequence)-1;
        if(length > MAX_READ_LENGTH) length = MAX_READ_LENGTH;

//        printf("%d, ", length);

        memcpy(&hReadSequence[n*MAX_READ_LENGTH], Read.sequence, length);
        memcpy(&hReadQualstr[n*MAX_READ_LENGTH], Read.qualstr, length);
        hReadLen[n] = length;

        n++;
        if(n == max_num_to_read) break;

    }
    hNumberOfReads = hNumberOfReads + n;                           // update global variable to total number of reads
    return(n);
}

int WriteBlockOfReads(int num_of_reads_to_write, const short *hReadLen, short min_out_read_length)
{
    int n=0;
    int w=0;
    short length;
    char str_buffer[MAX_FASTQ_BUF_LINE];

    if(num_of_reads_to_write == 0) return(0);
    if(verbose) printf("I/O > num_of_reads_to_write = %d\n", num_of_reads_to_write);
    while(gzgets(templateFile, Read.name, MAX_FASTQ_BUF_LINE) != NULL)
    {
        if(gzgets(templateFile, Read.sequence, MAX_FASTQ_BUF_LINE)==NULL)
        {
            printf("ERROR parsing input file for writing trimmed output!\n");
            exit(255);
        }
        if(gzgets(templateFile, Read.descr, MAX_FASTQ_BUF_LINE)==NULL)
        {
            printf("ERROR parsing input file for writing trimmed output!\n");
            exit(255);
        }
        if(gzgets(templateFile, Read.qualstr, MAX_FASTQ_BUF_LINE)==NULL)
        {
            printf("ERROR parsing input file for writing trimmed output!\n");
            exit(255);
        }

        length = hReadLen[n];

        if(gRNATrimming)                         // isolate the gRNA from adaptor match
        {
        	if(length>0)
        	{
        		strncpy(str_buffer, &Read.sequence[length+gRNA_start], gRNA_length);
        		strncpy(&Read.sequence[0], str_buffer, gRNA_length);
        		Read.sequence[gRNA_length]=10;
        		Read.sequence[gRNA_length+1]=0;
        		strncpy(str_buffer, &Read.qualstr[length+gRNA_start], gRNA_length);
        		strncpy(&Read.qualstr[0], str_buffer, gRNA_length);
        		Read.qualstr[gRNA_length]=10;
        		Read.qualstr[gRNA_length+1]=0;

                fprintf(outputFile,"%s", Read.name);
                fprintf(outputFile,"%s", Read.sequence);
                fprintf(outputFile,"%s", Read.descr);
                fprintf(outputFile,"%s", Read.qualstr);

                w++;
        	}
        }
        else                                     // normal quality and adaptor trimming
        {
            if(length >= min_out_read_length)
            {
                Read.sequence[length]=10;        // write only reads that are longer than minimum to output file
                Read.sequence[length+1]=0;
                Read.qualstr[length]=10;
                Read.qualstr[length+1]=0;

                fprintf(outputFile,"%s", Read.name);
                fprintf(outputFile,"%s", Read.sequence);
                fprintf(outputFile,"%s", Read.descr);
                fprintf(outputFile,"%s", Read.qualstr);

                w++;
             }
        }

        n++;
        if(n == num_of_reads_to_write) break;
    }
    hReadsWritten = hReadsWritten + w;
    return(n);
}

void CloseFiles(void)
{
    gzclose(inputFile);
    gzclose(templateFile);
    if(write_gz_output || write_pigz_output) pclose(outputFile); else fclose(outputFile);
}

bool Detect_FastQ_Format_PHRED(void)
{
	int n = 0;                                                                    // returns the number of reads read
	short length;
	uint min_qual = 65;
	uint max_qual = 65;
	bool deciphered = false;

	inputFile = gzopen(input_filename, "r");
    if(inputFile == NULL)
    {
    	printf("ERROR opening input file %s\n", input_filename);
    	exit(255);
    }

    while(gzgets(inputFile, Read.name, MAX_FASTQ_BUF_LINE) != NULL)
    {
    	if(Read.name[0] == 13) break;				// end if CR <13> or LF <10> are encountered
    	if(Read.name[0] == 10) break;
    	if(gzgets(inputFile, Read.sequence, MAX_FASTQ_BUF_LINE)==NULL)
        {
            printf("ERROR parsing input!\n");
            exit(255);
        }
        if(gzgets(inputFile, Read.descr, MAX_FASTQ_BUF_LINE)==NULL)
        {
            printf("ERROR parsing input!\n");
            exit(255);
        }
        if(gzgets(inputFile, Read.qualstr, MAX_FASTQ_BUF_LINE)==NULL)
        {
            printf("ERROR parsing input!\n");
            exit(255);
        }

        if(!FastQ_WARNING)                         // write warning if FastQ file contains non-standard lines
        {
            if(Read.name[0]!='@' || Read.descr[0]!='+')
            {
            	printf("Warning: Encountered unusual line identifiers in FastQ file format.");
            	FastQ_WARNING = true;
            }
        }
        length = strlen(Read.sequence)-1;

        for(n=0; n<length;n++)
        {
        	if(min_qual > (uint)Read.qualstr[n])
        	{
        		min_qual = (uint) Read.qualstr[n];
        	}
        	else
        	{
            	if(max_qual < (uint)Read.qualstr[n]) max_qual = (uint) Read.qualstr[n];

        	}
        }
        if(min_qual < 59)
        {
        	PHRED = 33;                                    // FastQ Sanger and Illumina 1.8+ format
        	deciphered = true;
        	break;
        }
        if(max_qual > 80)
        {
        	PHRED = 64;                                    // FastQ Illumina 1.3+ format
        	deciphered = true;
        	break;
        }
    }

    gzclose(inputFile);
    return(deciphered);
}

int PreprocessReadsOnGPU(void)
{
	bool DONE = false;
	bool ALL_DONE = false;

	int ReadsProcessed = 0;
    int ReadsToProcess[MAX_GPU_NUM];
    int ReadsSentToGPU[MAX_GPU_NUM];
    int dev_side = 0;                             // left or right side of device memories [0] or [1]
    int dev_next_side = 1;
    int side_flip;

    if(verbose) printf("\nPreprocessing reads on GPU\n");


    for(int dev=0; dev<numGPUs; dev++)                         // copy data and run kernels initial round to fill CUDA queue and not waste time !
    {
    	ReadsToProcess[dev] = 0;
    	ReadsSentToGPU[dev] = 0;
    	if(!DONE)
        {
        	ReadsToProcess[dev] = ReadBlockOfReads(MAX_READS_ON_GPU, hReadLen[dev][dev_side], hReadSequence[dev][dev_side], hReadQualstr[dev]);

            if(ReadsToProcess[dev] < MAX_READS_ON_GPU)
            {
            	DONE = true;
            }
            if(ReadsToProcess[dev] == 0) continue;

    	    CUDA_CHECK_RETURN(hipSetDevice(device[dev]));

    	    if(QualityTrimming) CUDA_CHECK_RETURN(hipMemcpyAsync(dQualArray[dev], hReadQualstr[dev], ReadsToProcess[dev] * MAX_READ_LENGTH * sizeof(char), hipMemcpyHostToDevice, upload_stream[dev]));
    	    CUDA_CHECK_RETURN(hipMemcpyAsync(dIntArray[dev][dev_side], hReadLen[dev][dev_side], ReadsToProcess[dev] * sizeof(short), hipMemcpyHostToDevice, upload_stream[dev]));
    	    CUDA_CHECK_RETURN(hipEventRecord(QUAL_COPY_DONE[dev], upload_stream[dev]));

    	    CUDA_CHECK_RETURN(hipStreamWaitEvent(compute_stream[dev], QUAL_COPY_DONE[dev], 0));
    	    if(QualityTrimming)
    	    {
    	    	if(verbose) printf("[%d] TrimmQuals\n", dev);
    	    	TrimmQuals<<<4096, 256, 0, compute_stream[dev]>>>(dQualArray[dev], dIntArray[dev][dev_side], ReadsToProcess[dev], MAX_READ_LENGTH);
    	    	CUDA_CHECK_RETURN(hipGetLastError());
    	    }
    	    CUDA_CHECK_RETURN(hipEventRecord(TrimmQ_DONE[dev], compute_stream[dev]));

    	    if(AdaptorTrimming || gRNATrimming) CUDA_CHECK_RETURN(hipMemcpyAsync(dSequenceArray[dev], hReadSequence[dev][dev_side], ReadsToProcess[dev] * MAX_READ_LENGTH * sizeof(char), hipMemcpyHostToDevice, upload_stream[dev]));
    	    CUDA_CHECK_RETURN(hipEventRecord(SEQ_COPY_DONE[dev], upload_stream[dev]));

    	    CUDA_CHECK_RETURN(hipStreamWaitEvent(compute_stream[dev], SEQ_COPY_DONE[dev], 0));
    	    if(gRNATrimming)
    	    {
    	    	if(verbose) printf("[%d] Find_gRNA\n", dev);
    	        Find_gRNA<<<4096, 256, 0, compute_stream[dev]>>>(dSequenceArray[dev], dIntArray[dev][dev_side], ReadsToProcess[dev], MAX_READ_LENGTH);
    	        CUDA_CHECK_RETURN(hipGetLastError());
    	    }
    	    else
    	    {
    	    	if(AdaptorTrimming)
    	    	{
    	    		if(verbose) printf("[%d] TrimmAdaptors\n", dev);
    	    		TrimmAdaptors<<<4096, 256, 0, compute_stream[dev]>>>(dSequenceArray[dev], dIntArray[dev][dev_side], ReadsToProcess[dev], MAX_READ_LENGTH);
    	    		CUDA_CHECK_RETURN(hipGetLastError());
    	    	}
    	    }
    	    CUDA_CHECK_RETURN(hipEventRecord(TrimmA_DONE[dev], compute_stream[dev]));

    	    CUDA_CHECK_RETURN(hipStreamWaitEvent(download_stream[dev], TrimmA_DONE[dev], 0));
    	    CUDA_CHECK_RETURN(hipMemcpyAsync(hReadLen[dev][dev_side], dIntArray[dev][dev_side], ReadsToProcess[dev] * sizeof(short), hipMemcpyDeviceToHost, download_stream[dev]));
    	    CUDA_CHECK_RETURN(hipEventRecord(DTH_COPY_DONE[dev], download_stream[dev]));
    	    if(verbose) printf("[%d] Memcpy DtoH %d reads\n", dev, ReadsToProcess[dev]);

    	    ReadsSentToGPU[dev] = ReadsToProcess[dev];
    	    ReadsToProcess[dev] = 0;
       }
    }

    while(!ALL_DONE)
    {
    	ALL_DONE = true;                                            // setup for test if should stop if there is no more data on the GPU

    	if(!DONE)
        {
            for(int dev=0; dev<numGPUs; dev++)                         // read a block of data, copy QUALS to GPU and run QUAL kernel
        	{
            	ReadsToProcess[dev] = 0;

        	    CUDA_CHECK_RETURN(hipSetDevice(device[dev]));

            	if(ReadsSentToGPU[dev]>0)
            	{
            		CUDA_CHECK_RETURN(hipEventSynchronize(TrimmQ_DONE[dev]));
            	}

            	ReadsToProcess[dev] = ReadBlockOfReads(MAX_READS_ON_GPU, hReadLen[dev][dev_next_side], hReadSequence[dev][dev_next_side], hReadQualstr[dev]);
                if(ReadsToProcess[dev] < MAX_READS_ON_GPU)
                {
                	DONE = true;
                }
                if(ReadsToProcess[dev] == 0) break;

                ALL_DONE = false;                                      // there is more work on the GPUs that requires further processing

                if(QualityTrimming) CUDA_CHECK_RETURN(hipMemcpyAsync(dQualArray[dev], hReadQualstr[dev], ReadsToProcess[dev] * MAX_READ_LENGTH * sizeof(char), hipMemcpyHostToDevice, upload_stream[dev]));
        	    CUDA_CHECK_RETURN(hipMemcpyAsync(dIntArray[dev][dev_next_side], hReadLen[dev][dev_next_side], ReadsToProcess[dev] * sizeof(short), hipMemcpyHostToDevice, upload_stream[dev]));
        	    CUDA_CHECK_RETURN(hipEventRecord(QUAL_COPY_DONE[dev], upload_stream[dev]));

        	    CUDA_CHECK_RETURN(hipStreamWaitEvent(compute_stream[dev], QUAL_COPY_DONE[dev], 0));
        	    if(QualityTrimming)
        	    {
        	    	if(verbose) printf("[%d] TrimmQuals\n", dev);
        	    	TrimmQuals<<<4096, 256, 0, compute_stream[dev]>>>(dQualArray[dev], dIntArray[dev][dev_next_side], ReadsToProcess[dev], MAX_READ_LENGTH);
        	    	CUDA_CHECK_RETURN(hipGetLastError());
        	    }
        	}
        }

        for(int dev=0; dev<numGPUs; dev++)                         // copy sequence and run Adaptor trim kernel
    	{
        	if(ReadsToProcess[dev]>0)
        	{
        	    CUDA_CHECK_RETURN(hipSetDevice(device[dev]));

        	    if(ReadsSentToGPU[dev]>0)
        	    {
            	    CUDA_CHECK_RETURN(hipEventSynchronize(TrimmA_DONE[dev]));
        	    }

        	    if(AdaptorTrimming || gRNATrimming) CUDA_CHECK_RETURN(hipMemcpyAsync(dSequenceArray[dev], hReadSequence[dev][dev_next_side], ReadsToProcess[dev] * MAX_READ_LENGTH * sizeof(char), hipMemcpyHostToDevice, upload_stream[dev]));
        	    CUDA_CHECK_RETURN(hipEventRecord(SEQ_COPY_DONE[dev], upload_stream[dev]));

        	    CUDA_CHECK_RETURN(hipStreamWaitEvent(compute_stream[dev], SEQ_COPY_DONE[dev], 0));
        	    if(gRNATrimming)
        	    {
        	    	if(verbose) printf("[%d] Find_gRNA\n", dev);
        	    	Find_gRNA<<<4096, 256, 0, compute_stream[dev]>>>(dSequenceArray[dev], dIntArray[dev][dev_next_side], ReadsToProcess[dev], MAX_READ_LENGTH);
        	    	CUDA_CHECK_RETURN(hipGetLastError());
        	    }
        	    else
        	    {
        	    	if(AdaptorTrimming)
        	    	{
        	    		if(verbose) printf("[%d] TrimmAdaptors\n", dev);
        	    		TrimmAdaptors<<<4096, 256, 0, compute_stream[dev]>>>(dSequenceArray[dev], dIntArray[dev][dev_next_side], ReadsToProcess[dev], MAX_READ_LENGTH);
        	    		CUDA_CHECK_RETURN(hipGetLastError());
        	    	}
        	    }
        	    CUDA_CHECK_RETURN(hipEventRecord(TrimmA_DONE[dev], compute_stream[dev]));
        	}
    	}

        for(int dev=0; dev<numGPUs; dev++)                          // copy data from devices and write output file
    	{
            if(verbose) printf("[%d] > ReadsSentToGPU = %d\n", dev, ReadsSentToGPU[dev]);

        	if(ReadsSentToGPU[dev]>0)
        	{
        	    CUDA_CHECK_RETURN(hipSetDevice(device[dev]));
        		CUDA_CHECK_RETURN(hipEventSynchronize(DTH_COPY_DONE[dev]));
        		if(verbose) printf("[%d] Writing hReadLen[%d][%d]\n", dev, dev, dev_side);
        	    WriteBlockOfReads(ReadsSentToGPU[dev], hReadLen[dev][dev_side], MIN_OUT_READ_LEN);    // write output block
        	    ReadsProcessed += ReadsSentToGPU[dev];             // count how many reads were considered for writing - actual output has minimal length requirement
        	}

        	if(ReadsToProcess[dev]>0)
        	{
        	    CUDA_CHECK_RETURN(hipStreamWaitEvent(download_stream[dev], TrimmA_DONE[dev], 0));
        	    CUDA_CHECK_RETURN(hipMemcpyAsync(hReadLen[dev][dev_next_side], dIntArray[dev][dev_next_side], ReadsToProcess[dev] * sizeof(short), hipMemcpyDeviceToHost, download_stream[dev]));
        	    CUDA_CHECK_RETURN(hipEventRecord(DTH_COPY_DONE[dev], download_stream[dev]));
        	    if(verbose) printf("[%d] Memcpy DtoH %d reads\n", dev, ReadsToProcess[dev]);
        	}
    	    ReadsSentToGPU[dev] = ReadsToProcess[dev];
    	    ReadsToProcess[dev] = 0;
    	}
        side_flip = dev_side;                                      // dev_side alternate between 0 and 1 to use memory efficiently
        dev_side = dev_next_side;
        dev_next_side = side_flip;
    }
    return(ReadsProcessed);
}

void usage_help(void)
{
	printf("\nProgram Options:\n\n");
	printf("\t-i, -I ....... specify the path for the input file [required]\n");
	printf("\t-o, -O ....... specify the path for the output file [required]\n");
	printf("\t-len, -Len ... specify the minimum length for reads written to the output file\n");
	printf("\t-f ........... over write output file if it exists\n");
	printf("\t-gz .......... write output to file in compressed format (.gz)\n");
	printf("\t-pigz ........ write output to file in compressed format (.gz) using multithreaded compression\n");
	printf("\t-v, -V ....... verbose, prints details on processing to screen\n");
	printf("\t-a, -A ....... adaptor sequence, can contain Ns that will match any base in the read\n");
	printf("\t-mm, -MM ..... number of mismatches allowed in adaptor search (not including Ns)\n");
	printf("\t-Phred ....... sets the base of the qualities PHRED values [default 33], use AUTO for autodetection\n");
	printf("\t-Qmin ........ sets the minimum acceptable quality score for a good base [default 20]\n");
	printf("\t-Qualab ...... sets the minimum average base quality score for remainder of the read after a bad base [default 25]\n");
	printf("\t-dev, -Dev ... specify CUDA device to use (can be used repeatedly for selecting multiple GPUs,\n");
	printf("\t               if not specified an automatic search for GPUs with compute capability 3.5 and 6 GB memory is performed)\n");
	printf("\nAlgorithm selection:\n\n");
	printf("\t/TA .......... select adaptor trimming only (can be also used before option /gRNA to avoid quality trimming)\n");
	printf("\t/TQ .......... select quality trimming only\n");
	printf("\t/TAQ ......... select quality and adaptor trimming [default]\n");
	printf("\t/gRNA ........ select gRNA recovery, followed by start relative to start of adaptor sequence and length of gRNA to be retrieved\n");
	printf("\t               ( /gRNA 12 20 will retrieve 20 nucleotide read sequence starting 12 positions after the 5' end match of adaptor)\n");
	printf("\nProgrammer options variables:\n");
	printf("\t-G ........... set MAX_READS_ON_GPU variable\n");
	printf("\t-R ........... set MAX_READ_LENGTH variable\n");
	printf("\nexample usage:\n");
	printf("\tPreprocessReads -dev 0 -dev 1 -i infile.fastq -o outfilename.fastq -len 26 -a ATTAGATCGATGCTA -mm 2\n\n");
	exit(3);
}

void invalid_command_line_option(const char *errorstr)
{
	printf("\nERROR %s\n", errorstr);
	printf("\nUsage ----------------------------------------------\n");
	usage_help();
}

void parseCommandLineArguments(int argc, char *argv[])
{
	bool inputfile_set = false;
	bool outputfile_set = false;
	bool adaptor_set = false;
	bool mismatch_set = false;
	bool outlen_set = false;
	for(int n=1; n<argc; n++)
	{
		if(argv[n][0] == '-')
		{
			char a = argv[n][1];
			switch (a)
			{
				case 'I':
				case 'i':
					if(inputfile_set) invalid_command_line_option("option -i: input file name already specified.");
					if(++n<argc)
					{
						if(strlen(argv[n])>400) invalid_command_line_option("option -i: input file path length exceeds limit.");
						strcpy(input_filename, argv[n]);
						inputfile_set=true;
						break;
					}
					else invalid_command_line_option("option -i: INPUT file name missing.");
					break;
				case 'O':
				case 'o':
					if(outputfile_set) invalid_command_line_option("option -o: output file name already specified.");
					if(++n<argc)
					{
						if(strlen(argv[n])>400) invalid_command_line_option("option -o: output file path length exceeds limit.");
						strcpy(output_filename, argv[n]);
						outputfile_set=true;
						break;
					}
					else invalid_command_line_option("option -o: OUTPUT file name missing.");
					break;
				case 'l':
				case 'L':
					if(argv[n][2]!='e' || argv[n][3]!='n') invalid_command_line_option("invalid command line option encountered.");
					if(outlen_set) invalid_command_line_option("option -len: minimum read length to include in output file is already specified.");
					if(++n<argc)
					{
						int ol=atoi(argv[n]);
						if(ol<0 || ol>MAX_READ_LENGTH) invalid_command_line_option("option -len: encountered invalid minimum read length to include in output file.");
						MIN_OUT_READ_LEN = (short) ol;
						outlen_set=true;
						break;
					}
					else invalid_command_line_option("option -len: expected minimum output read length not found.");
					break;

				case 'A':
				case 'a':
					if(adaptor_set) invalid_command_line_option("option -a: adaptor sequence already specified.");
					if(++n<argc)
					{
						if(strlen(argv[n])>MAX_ADAPTOR_LENGTH) invalid_command_line_option("option -a: adaptor sequence length exceeds limit.");
						if(strlen(argv[n])<3) invalid_command_line_option("option -a: adaptor sequence length too short.");
						strcpy(Adaptor, argv[n]);
						adaptor_set=true;
						break;
					}
					else invalid_command_line_option("option -a: expected adaptor sequence.");
					break;
				case 'M':
				case 'm':
					if(argv[n][2]!='m' && argv[n][2]!='M') invalid_command_line_option("invalid command line option encountered.");
					if(mismatch_set) invalid_command_line_option("option -mm: number of mismatches is already specified.");
					if(++n<argc)
					{
						int mm=atoi(argv[n]);
						if(mm>20) invalid_command_line_option("option -mm: number of mismatches exceeds limit.");
						mismatches = (short) mm;
						mismatch_set=true;
						break;
					}
					else invalid_command_line_option("option -mm: expected number of mismatches not found.");
					break;
				case 'd':
				case 'D':
					if(argv[n][2]!='e' || argv[n][3]!='v') invalid_command_line_option("invalid command line option encountered.");
					if(numGPUs>=MAX_GPU_NUM) invalid_command_line_option("option -dev: number of CUDA devices exceeds maximum.");
					if(++n<argc)
					{
						int d=atoi(argv[n]);
						if(d<0 || d>20) invalid_command_line_option("option -dev: invalid device number.");
						device[numGPUs] = d;
						numGPUs++;
						break;
					}
					else invalid_command_line_option("option -dev: expected CUDA device number not found.");
					break;
				case 'f':
					overwrite_outputfile = true;                                        // -f
					break;
				case 'g':                                                               // -gz
					if((argv[n][2]=='z') && (strlen(argv[n])==3))
					{
						write_gz_output = true;
						break;
					}
					else invalid_command_line_option("invalid command line option encountered.");
					break;
				case 'p':
				case 'P':
					if((strcmp(argv[n], "-Pigz")==0) || (strcmp(argv[n], "-pigz")==0))  // -pigz
					{
						write_pigz_output = true;
						break;
					}
					else                                                                // -phred
					{
						if(argv[n][2]!='h' || argv[n][3]!='r' || argv[n][4]!='e' || argv[n][5]!='d') invalid_command_line_option("invalid command line option encountered.");
						if(++n<argc)
						{
							if((strcmp(argv[n], "AUTO")==0) || (strcmp(argv[n], "auto")==0))
							{
								auto_phred_detect = true;
								break;
							}
							else
							{
								int d=atoi(argv[n]);
								if(d<0 || d>100) invalid_command_line_option("option -Phred: invalid phred base.");
								PHRED = d;
								break;
							}
						}
						else invalid_command_line_option("option -phred: expected number or AUTO not found.");
					}
					break;
				case 'Q':
				case 'q':
					if((strcmp(argv[n], "-Qmin")==0) || (strcmp(argv[n], "-qmin")==0))
					{
						if(++n<argc)
						{
							int qm=atoi(argv[n]);
							if(qm<10 || qm>64) invalid_command_line_option("option -qmin: outside 10 to 64 limit.");
							MIN_BASE_QUAL = (short) qm;
							break;
						}
						else invalid_command_line_option("option -qmin: expected number not found.");
					}
					else
					{
						if((strcmp(argv[n], "-Qualab")==0) || (strcmp(argv[n], "-qualab")==0))
						{
							if(++n<argc)
							{
								int qm=atoi(argv[n]);
								if(qm<10 || qm>64) invalid_command_line_option("option -qualab: outside 10 to 64 limit.");
								MIN_QUAL_AFTER_BAD = (short) qm;
								break;
							}
							else invalid_command_line_option("option -qmin: expected number not found.");
						}
					}
					break;
				case 'v':
				case 'V':
					verbose = true;
					break;
				case 'H':
				case 'h':
				case '?':
					printf("\nHELP ----------------------------------------------\n");
					usage_help();
					break;
				case 'G':                                                           // programmer option to set MAX_NUM_READS_ON_GPU parameter
					if(++n<argc)
					{
						int mrg=atoi(argv[n]);
						if(mrg<=0) invalid_command_line_option("invalid command line option encountered.");
						MAX_READS_ON_GPU = mrg;
						break;
					}
					else invalid_command_line_option("invalid command line option encountered.");
					break;
				case 'R':                                                           // programmer option to set MAX_READ_LENGTH parameter
					if(++n<argc)
					{
						int mrl=atoi(argv[n]);
						if(mrl<=0 || mrl>10000) invalid_command_line_option("invalid command line option encountered.");
						MAX_READ_LENGTH = (short) mrl;
						break;
					}
					else invalid_command_line_option("invalid command line option encountered.");
					break;

				default:
					invalid_command_line_option("invalid command line option encountered.");
			}
		}
		else
		{
			if(argv[n][0] == '/')
			{
				char a = argv[n][1];
				switch (a)
				{
					case 'T':
						AdaptorTrimming = false;
						QualityTrimming = false;
						gRNATrimming = false;
						if(strlen(argv[n]) == 3)
						{
							if(argv[n][2]=='A')
							{
								AdaptorTrimming = true;
								break;
							}
							else
							{
								if(argv[n][2]=='Q')
								{
									QualityTrimming = true;
									break;
								}
								else invalid_command_line_option("option /T: algorithm selection is invalid.");
							}
						}
						else
						{
							if(strlen(argv[n])==3 && argv[n][2]=='A' && argv[n][3]=='Q')
							{
								AdaptorTrimming = true;
								QualityTrimming = true;
								break;
							}
							else invalid_command_line_option("option /T: algorithm selection is invalid.");
						}
						break;
					case 'g':
						if(strlen(argv[n]) == 5 && argv[n][2]=='R' && argv[n][3]=='N' && argv[n][4]=='A')
						{
							gRNATrimming = true;
							if(++n < argc)
							{
								gRNA_start = atoi(argv[n]);
							}
							else invalid_command_line_option("option /gRNA: expected start position relative to adaptor 5' not found.");
							if(++n < argc)
							{
								gRNA_length = atoi(argv[n]);
							}
							else invalid_command_line_option("option /gRNA: expected gRNA length not found.");
							break;
						}
						else invalid_command_line_option("option /g: algorithm selection is invalid.");
						break;
				}
			}
			else invalid_command_line_option("parsing command line options.");
		}
	}
	if(!inputfile_set || !outputfile_set || ((AdaptorTrimming || gRNATrimming) && !adaptor_set)) invalid_command_line_option("missing required command line options.");
	if(strcmp(input_filename, output_filename)==0)
	{
		printf("ERROR: encountered identical input and output file paths!\n");         // input file cannot be identical to output file
		exit(253);
	}
	printf("Setting parameters:\n");
	printf("\tInput file name  : %s\n", input_filename);
	printf("\tOutput file name : %s\n", output_filename);
	if(AdaptorTrimming || gRNATrimming) printf("\tAdaptor sequence : %s\n", Adaptor);
	if(verbose)
	{
		if(AdaptorTrimming || gRNATrimming) printf("\tMismatches = %d\n", mismatches);
		printf("\tTrimming: ");
		if(QualityTrimming) printf("Quality ");
		if(AdaptorTrimming) printf("Adaptors ");
		if(gRNATrimming) printf("Find gRNA");
		printf("\n");
		if(QualityTrimming)
		{
			printf("\tMinimum accepted base quality = %d; average quality of read after bad base = %d\n", MIN_BASE_QUAL, MIN_QUAL_AFTER_BAD);
		}
		printf("\tFastQ Quality Encoding: ");
		if(!auto_phred_detect) printf("PHRED+%d\n", PHRED); else printf("<AUTO DETECTION>\n");
		printf("\tMAX_READ_LENGTH  : %d\n", MAX_READ_LENGTH);
		printf("\tMinimum read length for output = %d\n", MIN_OUT_READ_LEN);
		if(overwrite_outputfile) printf("\tForce overwrite output file if it exists.\n");
	}
	if(numGPUs>0)
	{
		printf("\nUsing CUDA devices: ");
		for(int n=0; n<numGPUs; n++) printf("[%d]", device[n]);
		printf("\n");
	}
}

int main(int argc, char *argv[])
{
	time_t start_t = time(NULL);                        // for timing the process
	time_t end_t;
    int reads_processed = 0;

    parseCommandLineArguments(argc, argv);                        // get parameters from the command line
    if(numGPUs == 0)
    {
        if(FindGPUs() == 0)                             // check for GPUs
        {
            printf("\nERROR: No suitable CUDA devices found ... exiting\n");
            exit(1);
        }
        else
        {
            if(numGPUs == 1)
            	printf("\nOne CUDA device with at least compute capability 3.5 and 6 GB memory found.\n");
            else
                printf("\n%d CUDA devices with at least compute capability 3.5 and 6 GB memory found.\n", numGPUs);
        }
    }

    if(auto_phred_detect)
    {
    	if(Detect_FastQ_Format_PHRED())
    	{
    		if(verbose) printf("<AUTO DETECTION> FastQ Quality Encoding: PHRED+%d\n", PHRED);
    	}
    	else
    	{
    		printf("\nERROR: Unable to detect FastQ format version.\n");
    		exit(32);
    	}
    }

    OpenFiles();                                    // access files
    InitGPUs();
    SetAdaptorSequence(Adaptor, mismatches, PHRED, MIN_BASE_QUAL, MIN_QUAL_AFTER_BAD);

    reads_processed = PreprocessReadsOnGPU();

    printf("\nReads processed: %d\n", reads_processed);
    printf("Reads in input file %s : %d\n",input_filename, hNumberOfReads);
    printf("Reads written to output file %s : %d\n", output_filename, hReadsWritten);
    if(FastQ_WARNING) printf("Encountered potential issues with the FastQ file format in %s\n", input_filename);

    CloseFiles();
    FreeGPUs();

    end_t = time(NULL);
    printf("\nTotal time elapsed: %.0lf seconds\n", difftime(end_t, start_t));
//    printf("\n<Press RTN key to exit>\n");
//    getchar();

    return(0);
}
